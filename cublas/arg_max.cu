
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>
#include <hipblas.h>

#define CHECK_CUDA(call)                                                       \
    do                                                                         \
    {                                                                          \
        hipError_t status_ = call;                                            \
        if (status_ != hipSuccess)                                            \
        {                                                                      \
            fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
                    hipGetErrorName(status_), hipGetErrorString(status_));   \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    } while (0)

#define CHECK_CUBLAS(call)                                                              \
    do                                                                                  \
    {                                                                                   \
        hipblasStatus_t status_ = call;                                                  \
        if (status_ != HIPBLAS_STATUS_SUCCESS)                                           \
        {                                                                               \
            fprintf(stderr, "CUBLAS error (%s:%d): %d\n", __FILE__, __LINE__, status_); \
            exit(EXIT_FAILURE);                                                         \
        }                                                                               \
    } while (0)

int argmax(float *prob, int n)
{
    int max_i = 0;
    float max_p = prob[0];
    for (int i = 1; i < n; i++)
    {
        if (prob[i] > max_p)
        {
            max_i = i;
            max_p = prob[i];
        }
    }
    return max_i;
}

int argmax_cublas(float *prob, int n)
{
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    int res;
    CHECK_CUBLAS(hipblasIsamax(handle, n, prob, 1, &res));

    CHECK_CUBLAS(hipblasDestroy(handle));
    return res;
}

// Function to generate random for array
void initializeArray(float *h_in, int size)
{
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(0.0f, 10.0f);

    for (int i = 0; i < size; i++)
    {
        h_in[i] = dis(gen);
    }
}

int main()
{
    float temperature = 1.0f;
    float topp = 0.9f;
    unsigned long long rng_state = 314028;
    int vocab_size = 10000;

    // host allocation
    float *h_in = new float[vocab_size];
    int h_next = 0;
    initializeArray(h_in, vocab_size);
    h_next = argmax(h_in, vocab_size);
    std::cout << "h_next: " << h_next << std::endl;

    // device allocation
    float *d_logits;
    CHECK_CUDA(hipMalloc((void **)&d_logits, vocab_size * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_logits, h_in, vocab_size * sizeof(float), hipMemcpyHostToDevice));

    int d_next = argmax_cublas(d_logits, vocab_size);
    std::cout << "d_next: " << d_next << std::endl;

    return 0;
}
